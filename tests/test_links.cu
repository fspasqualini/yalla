#include "hip/hip_runtime.h"
#include "../include/dtypes.cuh"
#include "../include/links.cuh"
#include "../include/solvers.cuh"
#include "minunit.cuh"


__device__ float3 no_pw_int(float3 Xi, float3 r, float dist, int i, int j)
{
    float3 dF{0};
    return dF;
}


const char* square_of_four()
{
    Solution<float3, Tile_solver> points{4};
    Links links{4};
    auto forces = [&links](const float3* __restrict__ d_X, float3* d_dX) {
        return link_forces(links, d_X, d_dX);
    };

    // clang-format off
    points.h_X[0].x = 1;  points.h_X[0].y = 1;  points.h_X[0].z = 0;
    points.h_X[1].x = 1;  points.h_X[1].y = -1; points.h_X[1].z = 0;
    points.h_X[2].x = -1; points.h_X[2].y = -1; points.h_X[2].z = 0;
    points.h_X[3].x = -1; points.h_X[3].y = 1;  points.h_X[3].z = 0;
    points.copy_to_device();
    links.h_link[0].a = 0; links.h_link[0].b = 1;
    links.h_link[1].a = 1; links.h_link[1].b = 2;
    links.h_link[2].a = 2; links.h_link[2].b = 3;
    links.h_link[3].a = 3; links.h_link[3].b = 0;
    // clang-format on
    links.copy_to_device();

    auto com_i = center_of_mass(points);
    for (auto i = 0; i < 500; i++) { points.take_step<no_pw_int>(0.1, forces); }

    points.copy_to_host();
    auto com_f = center_of_mass(points);
    MU_ASSERT("Momentum in square", isclose(com_i.x, com_f.x));
    MU_ASSERT("Momentum in square", isclose(com_i.y, com_f.y));
    MU_ASSERT("Momentum in square", isclose(com_i.z, com_f.z));

    MU_ASSERT("Not close in x", isclose(points.h_X[0].x, points.h_X[1].x));
    MU_ASSERT("Not close in y", isclose(points.h_X[1].y, points.h_X[2].y));
    MU_ASSERT("Not close in z", isclose(points.h_X[2].z, points.h_X[3].z));

    return NULL;
}


const char* all_tests()
{
    MU_RUN_TEST(square_of_four);
    return NULL;
}

MU_RUN_SUITE(all_tests);
