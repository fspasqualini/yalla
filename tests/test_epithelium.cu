#include "hip/hip_runtime.h"
#include "minunit.cuh"
#include "../lib/dtypes.cuh"
#include "../lib/solvers.cuh"
#include "../lib/epithelium.cuh"

__device__ __managed__ Solution<pocell, 4, N2nSolver> X;


__device__ pocell epithelium(pocell Xi, pocell Xj, int i, int j) {
    pocell dF = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    if (i == j) return dF;

    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > 1) return dF;

    float F = 2*(0.6 - dist)*(1 - dist) + powf(1 - dist, 2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;

    dF = dF + polarity_force(Xi, Xj)*0.2;

    assert(dF.x == dF.x);  // For NaN f != f.
    return dF;
}

__device__ __managed__ nhoodint<pocell> potential = epithelium;


const char* test_line_of_four() {
    for (int i = 0; i < 4; i++) {
        X[i].x = 0.733333*cosf((i - 0.5)*M_PI/3);
        X[i].y = 0.733333*sinf((i - 0.5)*M_PI/3);
        X[i].z = 0;
        X[i].phi = (i - 0.5)*M_PI/3;
        X[i].theta = M_PI/2;
    }
    for (int i = 0; i < 250; i++) {
        X.step(1, potential, 4);
    }
    for (int i = 1; i < 4; i++) {
        float prod = sinf(X[0].theta)*sinf(X[i].theta)*cosf(X[0].phi - X[i].phi)
            + cosf(X[0].theta)*cosf(X[i].theta);
        mu_assert("ERROR: Polarity not aligned", mu_isclose(prod, 1));
    }
    return NULL;
}

const char* all_tests() {
    mu_run_test(test_line_of_four);
    return NULL;
}

mu_run_suite(all_tests)
