#include "hip/hip_runtime.h"
#include "../lib/dtypes.cuh"
#include "../lib/solvers.cuh"
#include "../lib/protrusions.cuh"
#include "minunit.cuh"


Solution<float3, 4, N2nSolver> bolls;
Protrusions<4> links;


__device__ float3 no_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF {0};
    return dF;
}

__device__ auto d_no_interaction = &no_interaction;
auto h_no_interaction = get_device_object(d_no_interaction, 0);

void link_forces(const float3* __restrict__ d_X, float3* d_dX) {
    link_force<<<(4 + 32 - 1)/32, 32>>>(d_X, d_dX, links.d_cell_id, 4);
}


const char* square_of_four() {
    bolls.h_X[0].x = 1;  bolls.h_X[0].y = 1;  bolls.h_X[0].z = 0;
    bolls.h_X[1].x = 1;  bolls.h_X[1].y = -1; bolls.h_X[1].z = 0;
    bolls.h_X[2].x = -1; bolls.h_X[2].y = -1; bolls.h_X[2].z = 0;
    bolls.h_X[3].x = -1; bolls.h_X[3].y = 1;  bolls.h_X[3].z = 0;
    bolls.memcpyHostToDevice();
    links.h_cell_id[0].a = 0; links.h_cell_id[0].b = 1;
    links.h_cell_id[1].a = 1; links.h_cell_id[1].b = 2;
    links.h_cell_id[2].a = 2; links.h_cell_id[2].b = 3;
    links.h_cell_id[3].a = 3; links.h_cell_id[3].b = 0;
    links.memcpyHostToDevice();

    auto com_i = center_of_mass(bolls);
    for (auto i = 0; i < 500; i++) {
        bolls.step(0.1, h_no_interaction, link_forces);
    }

    bolls.memcpyDeviceToHost();
    auto com_f = center_of_mass(bolls);
    MU_ASSERT("Momentum in square", MU_ISCLOSE(com_i.x, com_f.x));
    MU_ASSERT("Momentum in square", MU_ISCLOSE(com_i.y, com_f.y));
    MU_ASSERT("Momentum in square", MU_ISCLOSE(com_i.z, com_f.z));

    MU_ASSERT("Not close to origin in x", MU_ISCLOSE(bolls.h_X[0].x, 0));
    MU_ASSERT("Not close to origin in y", MU_ISCLOSE(bolls.h_X[0].y, 0));
    MU_ASSERT("Not close to origin in z", MU_ISCLOSE(bolls.h_X[0].z, 0));

    return NULL;
}


const char* all_tests() {
    MU_RUN_TEST(square_of_four);
    return NULL;
}

MU_RUN_SUITE(all_tests);
