#include "hip/hip_runtime.h"
#include "../include/dtypes.cuh"
#include "../include/solvers.cuh"
#include "../include/inits.cuh"
#include "minunit.cuh"


__device__ float4 oscillator(float4 Xi, float4 r, float dist, int i, int j) {
    float4 dF {0};
    if (i == j) return dF;

    if (i == 0) return Xi - r;

    return - (Xi - r);
}

const char* test_oscillation() {
    Solution<float4, 2, Tile_solver> oscillation;
    oscillation.h_X[0].w = 1;
    oscillation.h_X[1].w = 0;
    oscillation.copy_to_device();

    auto n_steps = 100;
    for (auto i = 0; i < n_steps; i++) {
        oscillation.take_step<oscillator>(2*M_PI/n_steps);
        oscillation.copy_to_host();
        MU_ASSERT("Oscillator off circle", MU_ISCLOSE(
            powf(oscillation.h_X[0].w, 2) + powf(oscillation.h_X[1].w, 2), 1));
    }
    oscillation.copy_to_host();
    MU_ASSERT("Oscillator final cosine", MU_ISCLOSE(oscillation.h_X[0].w, 1));
    // The sine is substantially less precise ;-)

    return NULL;
}


const auto L_0 = 0.5;

__device__ float3 clipped_spring(float3 Xi, float3 r, float dist, int i, int j) {
    float3 dF {0};
    if (i == j) return dF;

    if (dist >= 1) return dF;

    dF = r*(L_0 - dist)/dist;
    return dF;
}

const char* test_tile_tetrahedron() {
    Solution<float3, 4, Tile_solver> tile;
    uniform_sphere(L_0, tile);
    auto com_i = center_of_mass(tile);
    for (auto i = 0; i < 500; i++) {
        tile.take_step<clipped_spring>(0.1);
    }

    tile.copy_to_host();
    for (auto i = 1; i < 4; i++) {
        auto r = tile.h_X[0] - tile.h_X[i];
        auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        MU_ASSERT("Spring not relaxed in tile tetrahedron", MU_ISCLOSE(dist, L_0));
    }

    auto com_f = center_of_mass(tile);
    MU_ASSERT("Momentum in tile tetrahedron", MU_ISCLOSE(com_i.x, com_f.x));
    MU_ASSERT("Momentum in tile tetrahedron", MU_ISCLOSE(com_i.y, com_f.y));
    MU_ASSERT("Momentum in tile tetrahedron", MU_ISCLOSE(com_i.z, com_f.z));

    return NULL;
}

const char* test_grid_tetrahedron() {
    Solution<float3, 4, Grid_solver> grid;
    uniform_sphere(L_0, grid);
    auto com_i = center_of_mass(grid);
    for (auto i = 0; i < 500; i++) {
        grid.take_step<clipped_spring>(0.1);
    }

    grid.copy_to_host();
    for (auto i = 1; i < 4; i++) {
        auto r = float3{grid.h_X[0].x - grid.h_X[i].x, grid.h_X[0].y - grid.h_X[i].y,
            grid.h_X[0].z - grid.h_X[i].z};
        auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        MU_ASSERT("Spring not relaxed in grid tetrahedron", MU_ISCLOSE(dist, L_0));
    }

    auto com_f = center_of_mass(grid);
    MU_ASSERT("Momentum in grid tetrahedron", MU_ISCLOSE(com_i.x, com_f.x));
    MU_ASSERT("Momentum in grid tetrahedron", MU_ISCLOSE(com_i.y, com_f.y));
    MU_ASSERT("Momentum in grid tetrahedron", MU_ISCLOSE(com_i.z, com_f.z));

    return NULL;
}

const auto n_max = 50;

const char* test_compare_methods() {
    Solution<float3, n_max, Tile_solver> tile;
    Solution<float3, n_max, Grid_solver> grid;
    uniform_sphere(0.733333, tile);
    for (auto i = 0; i < n_max; i++) {
        grid.h_X[i].x = tile.h_X[i].x;
        grid.h_X[i].y = tile.h_X[i].y;
        grid.h_X[i].z = tile.h_X[i].z;
    }
    grid.copy_to_device();
    for (auto i = 0; i < 2; i++) tile.take_step<clipped_spring>(0.5);
    for (auto i = 0; i < 2; i++) grid.take_step<clipped_spring>(0.5);

    tile.copy_to_host();
    grid.copy_to_host();
    for (auto i = 0; i < n_max; i++) {
        MU_ASSERT("Methods disagree", MU_ISCLOSE(tile.h_X[i].x, grid.h_X[i].x));
        MU_ASSERT("Methods disagree", MU_ISCLOSE(tile.h_X[i].y, grid.h_X[i].y));
        MU_ASSERT("Methods disagree", MU_ISCLOSE(tile.h_X[i].z, grid.h_X[i].z));
    }

    return NULL;
}


__device__ float3 no_pw_int(float3 Xi, float3 r, float dist, int i, int j) {
    return float3 {0};
}

__global__ void push_cell(float3* d_dX) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i != 0) return;

    d_dX[1] = float3{1, 0, 0};
}

void push(const float3* __restrict__ d_X, float3* d_dX) {
    push_cell<<<1, 1>>>(d_dX);
}

const char* test_generic_forces() {
    Solution<float3, 2, Tile_solver> tile;
    tile.h_X[0] = float3{0, 0, 10};
    tile.h_X[1] = float3{0, 0, 0};
    tile.copy_to_device();
    auto com_i = center_of_mass(tile);
    tile.take_step<no_pw_int>(1, push);

    tile.copy_to_host();
    auto com_f = center_of_mass(tile);
    MU_ASSERT("Momentum in tile generic force", MU_ISCLOSE(com_i.x, com_f.x));
    MU_ASSERT("Momentum in tile generic force", MU_ISCLOSE(com_i.y, com_f.y));
    MU_ASSERT("Momentum in tile generic force", MU_ISCLOSE(com_i.z, com_f.z));

    MU_ASSERT("Tile generic force failed in x", MU_ISCLOSE(tile.h_X[1].x, 0.5));
    MU_ASSERT("Tile generic force failed in y", MU_ISCLOSE(tile.h_X[1].y, 0));
    MU_ASSERT("Tile generic force failed in z", MU_ISCLOSE(tile.h_X[1].z, 0));

    Solution<float3, 2, Grid_solver> grid;
    grid.h_X[0] = float3{0, 0, 10};
    grid.h_X[1] = float3{0, 0, 0};
    grid.copy_to_device();
    com_i = center_of_mass(grid);
    grid.take_step<clipped_spring>(1, push);

    grid.copy_to_host();
    com_f = center_of_mass(grid);
    MU_ASSERT("Momentum in grid generic force", MU_ISCLOSE(com_i.x, com_f.x));
    MU_ASSERT("Momentum in grid generic force", MU_ISCLOSE(com_i.y, com_f.y));
    MU_ASSERT("Momentum in grid generic force", MU_ISCLOSE(com_i.z, com_f.z));

    MU_ASSERT("Grid generic force failed in x", MU_ISCLOSE(grid.h_X[1].x, 0.5));
    MU_ASSERT("Grid generic force failed in y", MU_ISCLOSE(grid.h_X[1].y, 0));
    MU_ASSERT("Grid generic force failed in z", MU_ISCLOSE(grid.h_X[1].z, 0));

    return NULL;
}


__device__ float global_friction(float3 Xi, float3 r, float dist, int i, int j) {
    return 0;
}

const char* test_friction() {
    Solution<float3, 2, Tile_solver> tile;
    tile.h_X[0] = float3{0,  0, 0};
    tile.h_X[1] = float3{.5, 0, 0};
    tile.copy_to_device();
    for (auto i = 0; i < 10; i++) tile.take_step<no_pw_int, global_friction>(0.05, push);
    tile.copy_to_host();
    MU_ASSERT("Tile global friction", MU_ISCLOSE(tile.h_X[1].x - tile.h_X[0].x, 1));

    tile.h_X[0] = float3{0,  0, 0};
    tile.h_X[1] = float3{.5, 0, 0};
    tile.copy_to_device();
    for (auto i = 0; i < 10; i++) tile.take_step<no_pw_int>(0.05, push);
    tile.copy_to_host();
    MU_ASSERT("Tile local friction", MU_ISCLOSE(tile.h_X[1].x - tile.h_X[0].x, 0.75));

    Solution<float3, 2, Grid_solver> grid;
    grid.h_X[0] = float3{0,  0, 0};
    grid.h_X[1] = float3{.5, 0, 0};
    grid.copy_to_device();
    for (auto i = 0; i < 10; i++) grid.take_step<no_pw_int, global_friction>(0.05, push);
    grid.copy_to_host();
    MU_ASSERT("Grid global friction", MU_ISCLOSE(grid.h_X[1].x - grid.h_X[0].x, 1));

    grid.h_X[0] = float3{0,  0, 0};
    grid.h_X[1] = float3{.5, 0, 0};
    grid.copy_to_device();
    for (auto i = 0; i < 10; i++) grid.take_step<no_pw_int>(0.05, push);
    grid.copy_to_host();
    MU_ASSERT("Grid local friction", MU_ISCLOSE(grid.h_X[1].x - grid.h_X[0].x, 0.75));

    return NULL;
}


__global__ void single_grid(const int* __restrict__ d_cube_id) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= 1000) return;

    auto expected_cube = (GRID_SIZE*GRID_SIZE*GRID_SIZE)/2
        + (GRID_SIZE*GRID_SIZE)/2 + GRID_SIZE/2
        + i%10 + (i%100/10)*GRID_SIZE + (i/100)*GRID_SIZE*GRID_SIZE;
    D_ASSERT(d_cube_id[i] == expected_cube);
}

__global__ void double_grid(const int* __restrict__ d_cube_id) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= 1000 - 8) return;

    D_ASSERT(d_cube_id[i] == d_cube_id[i - i%8]);
}

const char* test_grid_spacing() {
    Solution<float3, 1000, Grid_solver> bolls;
    for (auto i = 0; i < 10; i++) {
        for (auto j = 0; j < 10; j++) {
            for (auto k = 0; k < 10; k++) {
                bolls.h_X[100*i + 10*j + k].x = k + 0.5;
                bolls.h_X[100*i + 10*j + k].y = j + 0.5;
                bolls.h_X[100*i + 10*j + k].z = i + 0.5;
            }
        }
    }
    bolls.copy_to_device();

    Grid<1000> grid;
    grid.build(bolls, 1);
    single_grid<<<256, 4>>>(grid.d_cube_id);

    grid.build(bolls, 2);
    double_grid<<<256, 4>>>(grid.d_cube_id);
    hipDeviceSynchronize();  // Wait for device to exit

    return NULL;
}


const char* all_tests() {
    MU_RUN_TEST(test_oscillation);
    MU_RUN_TEST(test_tile_tetrahedron);
    MU_RUN_TEST(test_grid_tetrahedron);
    MU_RUN_TEST(test_compare_methods);
    MU_RUN_TEST(test_generic_forces);
    MU_RUN_TEST(test_friction);
    MU_RUN_TEST(test_grid_spacing);
    return NULL;
}

MU_RUN_SUITE(all_tests);
