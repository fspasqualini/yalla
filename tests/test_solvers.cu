#include "hip/hip_runtime.h"
#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "minunit.cuh"


const auto n_max = 1000;
const auto L_0 = 0.5;


__device__ float3 pairwise_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > 1) return dF;

    dF = r*(L_0 - dist)/dist;  // Spring force
    return dF;
}

#include "../lib/solvers.cuh"

Solution<float3, n_max, N2n_solver> n2n;
Solution<float3, n_max, Lattice_solver> latt;


const char* test_n2n_tetrahedron() {
    *n2n.h_n = 4;
    uniform_sphere(L_0, n2n);
    auto com_i = center_of_mass(n2n);
    for (auto i = 0; i < 500; i++) {
        n2n.take_step(0.1);
    }

    n2n.copy_to_host();
    for (auto i = 1; i < 4; i++) {
        auto r = n2n.h_X[0] - n2n.h_X[i];
        auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        MU_ASSERT("Spring not relaxed in n2n tetrahedron", MU_ISCLOSE(dist, L_0));
    }

    auto com_f = center_of_mass(n2n);
    MU_ASSERT("Momentum in n2n tetrahedron", MU_ISCLOSE(com_i.x, com_f.x));
    MU_ASSERT("Momentum in n2n tetrahedron", MU_ISCLOSE(com_i.y, com_f.y));
    MU_ASSERT("Momentum in n2n tetrahedron", MU_ISCLOSE(com_i.z, com_f.z));

    return NULL;
}

const char* test_latt_tetrahedron() {
    *latt.h_n = 4;
    uniform_sphere(L_0, latt);
    auto com_i = center_of_mass(latt);
    for (auto i = 0; i < 500; i++) {
        latt.take_step(0.1);
    }

    latt.copy_to_host();
    for (auto i = 1; i < 4; i++) {
        auto r = float3{latt.h_X[0].x - latt.h_X[i].x, latt.h_X[0].y - latt.h_X[i].y,
            latt.h_X[0].z - latt.h_X[i].z};
        auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        MU_ASSERT("Spring not relaxed in lattice tetrahedron", MU_ISCLOSE(dist, L_0));
    }

    auto com_f = center_of_mass(latt);
    MU_ASSERT("Momentum in lattice tetrahedron", MU_ISCLOSE(com_i.x, com_f.x));
    MU_ASSERT("Momentum in lattice tetrahedron", MU_ISCLOSE(com_i.y, com_f.y));
    MU_ASSERT("Momentum in lattice tetrahedron", MU_ISCLOSE(com_i.z, com_f.z));

    return NULL;
}


const char* test_compare_methods() {
    *n2n.h_n = n_max;
    *latt.h_n = n_max;
    uniform_sphere(0.733333, n2n);
    for (auto i = 0; i < n_max; i++) {
        latt.h_X[i].x = n2n.h_X[i].x;
        latt.h_X[i].y = n2n.h_X[i].y;
        latt.h_X[i].z = n2n.h_X[i].z;
    }
    latt.copy_to_device();
    n2n.take_step(0.5);
    latt.take_step(0.5);

    n2n.copy_to_host();
    latt.copy_to_host();
    for (auto i = 0; i < n_max; i++) {
        MU_ASSERT("Methods disagree", MU_ISCLOSE(n2n.h_X[i].x, latt.h_X[i].x));
        MU_ASSERT("Methods disagree", MU_ISCLOSE(n2n.h_X[i].y, latt.h_X[i].y));
        MU_ASSERT("Methods disagree", MU_ISCLOSE(n2n.h_X[i].z, latt.h_X[i].z));
    }

    return NULL;
}


__global__ void push(float3* d_dX) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= 1) return;

    d_dX[0] = float3{1, 0, 0};
}

void push_genforce(const float3* __restrict__ d_X, float3* d_dX) {
    push<<<1, 1>>>(d_dX);
}

const char* test_generic_forces() {
    n2n.h_X[0] = float3{0, 0, 0};
    n2n.copy_to_device();
    n2n.take_step(1, push_genforce);

    n2n.copy_to_host();
    MU_ASSERT("N2n Generic force failed", MU_ISCLOSE(n2n.h_X[0].x, 1));
    MU_ASSERT("N2n Generic force failed", MU_ISCLOSE(n2n.h_X[0].y, 0));
    MU_ASSERT("N2n Generic force failed", MU_ISCLOSE(n2n.h_X[0].z, 0));

    latt.h_X[0] = float3{0, 0, 0};
    latt.copy_to_device();
    latt.take_step(1, push_genforce);

    latt.copy_to_host();
    MU_ASSERT("Lattice Generic force failed", MU_ISCLOSE(latt.h_X[0].x, 1));
    MU_ASSERT("Lattice Generic force failed", MU_ISCLOSE(latt.h_X[0].y, 0));
    MU_ASSERT("Lattice Generic force failed", MU_ISCLOSE(latt.h_X[0].z, 0));

    return NULL;
}


template<int n_max>
__global__ void single_lattice(const Lattice<n_max>* __restrict__ d_lattice) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= 1000) return;

    auto expected_cube = (LATTICE_SIZE*LATTICE_SIZE*LATTICE_SIZE)/2
        + (LATTICE_SIZE*LATTICE_SIZE)/2 + LATTICE_SIZE/2
        + i%10 + (i%100/10)*LATTICE_SIZE + (i/100)*LATTICE_SIZE*LATTICE_SIZE;
    D_ASSERT(d_lattice->d_cube_id[i] == expected_cube);
}

template<int n_max>
__global__ void double_lattice(const Lattice<n_max>* __restrict__ d_lattice) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= 1000 - 8) return;

    D_ASSERT(d_lattice->d_cube_id[i] == d_lattice->d_cube_id[i - i%8]);
}

const char* test_lattice_spacing() {
    for (auto i = 0; i < 10; i++) {
        for (auto j = 0; j < 10; j++) {
            for (auto k = 0; k < 10; k++) {
                latt.h_X[100*i + 10*j + k].x = k + 0.5;
                latt.h_X[100*i + 10*j + k].y = j + 0.5;
                latt.h_X[100*i + 10*j + k].z = i + 0.5;
            }
        }
    }
    latt.copy_to_device();

    latt.build_lattice(1);
    single_lattice<<<256, 4>>>(latt.d_lattice);

    latt.build_lattice(2);
    double_lattice<<<256, 4>>>(latt.d_lattice);
    hipDeviceSynchronize();  // Wait for device to exit

    return NULL;
}


const char* all_tests() {
    MU_RUN_TEST(test_n2n_tetrahedron);
    MU_RUN_TEST(test_latt_tetrahedron);
    MU_RUN_TEST(test_compare_methods);
    MU_RUN_TEST(test_generic_forces);
    MU_RUN_TEST(test_lattice_spacing);
    return NULL;
}

MU_RUN_SUITE(all_tests);
