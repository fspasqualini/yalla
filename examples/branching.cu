#include "hip/hip_runtime.h"
// This program simulates a branching mechanism on a spheric organoid.
// A Turing mechanism taking place on its surface creates a pattern,
// peaks of activator induce local proliferation on the underlying cells,
// resulting on the growth of a branch.
#include <string>
#include <hiprand/hiprand_kernel.h>

#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/solvers.cuh"
#include "../include/vtk.cuh"
#include "../include/polarity.cuh"
#include "../include/property.cuh"
#include "../include/links.cuh"


const auto r_max_epi = 1.5;
const auto r_max_mes = 1.0;
const auto r_min_homotypic = 0.6;
const auto r_min_heterotypic = 0.8;
const auto skip_steps = 100;
const auto lambda = 1.;

// Turing parameters
const auto D_u = 0.05;
const auto D_v = 2.0;
const auto f_v = 1.0;
const auto f_u = 80.0;
const auto g_u = 80.0;
const auto m_u = 0.25;  // degradation rates
const auto m_v = 0.75;
const auto s_u = 0.05;

const auto r_min_min = min(r_min_homotypic,r_min_heterotypic);
const auto r_max_max = max(r_max_epi,r_max_mes);
const auto dt = 0.05*r_min_min*r_min_min/D_v;

const auto epi_proliferation_rate = 0.000893;
const auto mes_proliferation_rate = 0.000493;

// threshold conc. of v that allows mesench. cells to divide
const auto prolif_threshold = 1600.0f;

const auto n_0 = 1000;
const auto n_max = 65000;

enum Cell_types {mesenchyme, epithelium};

__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;  // number of mesenchymal neighbours
__device__ int* d_epi_nbs;

MAKE_PT(Cell, theta, phi, u, v);


__device__ Cell epi_turing_mes_noturing(Cell Xi, Cell r, float dist, int i, int j) {
    Cell dF {0};

    // Meinhard equations
    if (i == j) {
        if (d_type[i] == epithelium) {
            dF.u = lambda*((f_u*Xi.u*Xi.u)/(1 + f_v*Xi.v) - m_u*Xi.u + s_u);
            dF.v = lambda*(g_u*Xi.u*Xi.u - m_v*Xi.v);

            // prevent negative values
            if(-dF.u > Xi.u) dF.u = 0.0f;
            if(-dF.v > Xi.v) dF.v = 0.0f;
        }
        return dF;
    }

    float r_max;
    float r_min;
    if (d_type[i] == d_type[j]) {
        r_min=r_min_homotypic;
        if (d_type[i]==epithelium) r_max = r_max_epi;
        else r_max = r_max_mes;
    } else {
        r_min = r_min_heterotypic;
        r_max = r_max_mes;
    }

    if (dist > r_max) return dF;

    auto F = 2*(r_min - dist)*(r_max - dist) + powf(r_max - dist, 2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;

    // Diffusion
    if(d_type[i] == epithelium && d_type[j] == epithelium) {
        dF.u = -D_u*r.u;
        dF.v = -D_v*r.v;

        if(-dF.u > Xi.u) dF.u=0.0f;
        if(-dF.v > Xi.v) dF.v=0.0f;

        dF += rigidity_force(Xi, r, dist)*0.2;
    } else {
        dF.v = -D_v*r.v;  // inhibitor diffuses towards the mesenchyme to induce proliferation
    }

    if (d_type[j] == epithelium) atomicAdd(&d_epi_nbs[i], 1);
    else atomicAdd(&d_mes_nbs[i],1);

    return dF;
}


__global__ void proliferate(float mean_distance, Cell* d_X, int* d_n_cells,
        hiprandState* d_state) {
    D_ASSERT(*d_n_cells*epi_proliferation_rate <= n_max);
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= *d_n_cells*(1 - epi_proliferation_rate)) return;  // Dividing new cells is problematic!

    switch (d_type[i]) {
        case mesenchyme: {
            if(d_X[i].v < prolif_threshold) return;
            auto r = hiprand_uniform(&d_state[i]);
            if (r > mes_proliferation_rate) return;
            break;
        }
        case epithelium: {
            if (d_epi_nbs[i] > 20) return;

            auto r = hiprand_uniform(&d_state[i]);
            if (r > epi_proliferation_rate) return;
        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto phi = hiprand_uniform(&d_state[i])*M_PI;
    auto theta = hiprand_uniform(&d_state[i])*2*M_PI;
    d_X[n].x = d_X[i].x + mean_distance/4*sinf(theta)*cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance/4*sinf(theta)*sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance/4*cosf(theta);
    d_X[n].u = d_X[i].u/2;
    d_X[i].u = d_X[i].u/2;
    d_X[n].v = d_X[i].v/2;
    d_X[i].v = d_X[i].v/2;
    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = d_X[i].phi;
    d_type[n] = d_type[i];
}


int main(int argc, char const *argv[]) {
    // Command line arguments
    // argv[1] : output file taking
    // argv[2] : number of time steps
    auto n_time_steps = std::stoi(argv[2]);

    // Initial state
    Solution<Cell, n_max, Grid_solver> bolls(n_0);
    uniform_sphere(0.5, bolls);
    Property<n_max, Cell_types> type;
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    for (auto i = 0; i < n_0; i++) {
        bolls.h_X[i].u = 0;
        bolls.h_X[i].v = 0;
        type.h_prop[i] = mesenchyme;
    }
    bolls.copy_to_device();
    type.copy_to_device();
    Property<n_max, int> n_mes_nbs;
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    Property<n_max, int> n_epi_nbs;
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

    // State for proliferations
    hiprandState *d_state;
    hipMalloc(&d_state, n_max*sizeof(hiprandState));
    setup_rand_states<<<(n_max + 128 - 1)/128, 128>>>(d_state, n_max);

    // Relax
    for (auto time_step = 0; time_step <= 500; time_step++) {
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_0, 0);
        bolls.take_step<epi_turing_mes_noturing>(dt/100);
    }

    // Find epithelium
    bolls.copy_to_host();
    n_mes_nbs.copy_to_host();
    for (auto i = 0; i < n_0; i++) {
        if (n_mes_nbs.h_prop[i] < 35) {
            type.h_prop[i] = epithelium;
            auto dist = sqrtf(bolls.h_X[i].x*bolls.h_X[i].x
                + bolls.h_X[i].y*bolls.h_X[i].y + bolls.h_X[i].z*bolls.h_X[i].z);
            bolls.h_X[i].theta = acosf(bolls.h_X[i].z/dist);
            bolls.h_X[i].phi = atan2(bolls.h_X[i].y, bolls.h_X[i].x);

            bolls.h_X[i].u = rand()/(RAND_MAX + 1.)/5 - 0.1;
            bolls.h_X[i].v = rand()/(RAND_MAX + 1.)/5 - 0.1;

        } else {
            bolls.h_X[i].theta = 0;
            bolls.h_X[i].phi = 0;
        }

    }
    bolls.copy_to_device();
    type.copy_to_device();

    // Relax again to let epithelium stabilise
    for (auto time_step = 0; time_step <= 100; time_step++) {
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_0, 0);
        bolls.take_step<epi_turing_mes_noturing>(dt);
    }

    // Integrate positions
    Vtk_output output(argv[1]);
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        bolls.copy_to_host();

        proliferate<<<(bolls.get_d_n() + 128 - 1)/128, 128>>>(0.75, bolls.d_X,
            bolls.d_n, d_state);
        bolls.build_grid(r_max_max);
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + bolls.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop, n_epi_nbs.d_prop + bolls.get_d_n(), 0);

        bolls.take_step<epi_turing_mes_noturing>(dt);
        if(time_step%skip_steps == 0) {
            output.write_positions(bolls);
            output.write_polarity(bolls);
            output.write_field(bolls, "u", &Cell::u);
            output.write_field(bolls, "v", &Cell::v);
        }
    }

    return 0;
}
