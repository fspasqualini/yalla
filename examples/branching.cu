#include "hip/hip_runtime.h"
// Simulates branching on a spheroid induced by Turing mechanism on surface
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <thread>

#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/links.cuh"
#include "../include/polarity.cuh"
#include "../include/property.cuh"
#include "../include/solvers.cuh"
#include "../include/vtk.cuh"

const auto n_0 = 500;
const auto n_max = 200000;
const auto r_max = 1.0f;
const auto n_time_steps = 400;
const auto skip_steps = 10;
const auto dt = 0.2f;

// Turing parameters
const auto lambda = 0.0075;

const auto D_u = 0.001;
const auto D_v = 0.2;
const auto f_v = 1.0;
const auto f_u = 80.0;
const auto g_u = 80.0;
const auto m_u = 0.25;  // degradation rates
const auto m_v = 0.75;
const auto s_u = 0.05;

const auto epi_proliferation_rate = 0.2;
const auto mes_proliferation_rate = 0.1;
// Threshold conc. of v that allows mesench. cells to divide
const auto prolif_threshold = 1150.0f;

enum Cell_types { mesenchyme, epithelium };

__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;  // number of mesenchymal neighbours
__device__ int* d_epi_nbs;

MAKE_PT(Cell, theta, phi, u, v);


__device__ Cell epi_turing_mes_noturing(
    Cell Xi, Cell r, float dist, int i, int j)
{
    Cell dF{0};

    // Meinhard equations
    if (i == j) {
        if (d_type[i] == epithelium) {
            dF.u = lambda *
                   ((f_u * Xi.u * Xi.u) / (1 + f_v * Xi.v) - m_u * Xi.u + s_u);
            dF.v = lambda * (g_u * Xi.u * Xi.u - m_v * Xi.v);

            // Prevent negative values
            if (-dF.u > Xi.u) dF.u = 0.0f;
            if (-dF.v > Xi.v) dF.v = 0.0f;
        }
        return dF;
    }

    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        F = fmaxf(0.7 - dist, 0) * 2 - fmaxf(dist - 0.8, 0);
    } else {
        F = fmaxf(0.8 - dist, 0) * 2 - fmaxf(dist - 0.9, 0);
    }
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    // Diffusion
    if (d_type[i] == epithelium && d_type[j] == epithelium) {
        dF.u = -D_u * r.u;
        dF.v = -D_v * r.v;

        if (-dF.u > Xi.u) dF.u = 0.0f;
        if (-dF.v > Xi.v) dF.v = 0.0f;

        dF += rigidity_force(Xi, r, dist) * 0.2;
    } else {
        dF.v = -D_v * r.v;  // Diffuses into mesenchyme to induce proliferation
    }

    if (d_type[j] == epithelium)
        atomicAdd(&d_epi_nbs[i], 1);
    else
        atomicAdd(&d_mes_nbs[i], 1);

    return dF;
}


__global__ void proliferate(
    float mean_distance, Cell* d_X, int* d_n_cells, hiprandState* d_state)
{
    D_ASSERT(*d_n_cells * epi_proliferation_rate <= n_max);
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *d_n_cells * (1 - epi_proliferation_rate))
        return;  // Dividing new cells is problematic!

    auto rnd = hiprand_uniform(&d_state[i]);
    switch (d_type[i]) {
        case mesenchyme: {
            if (d_X[i].v < prolif_threshold) return;

            if (rnd > mes_proliferation_rate) return;

            break;
        }
        case epithelium: {
            if (d_epi_nbs[i] > 10) return;

            if (d_mes_nbs[i] <= 0) return;

            if (rnd > epi_proliferation_rate) return;
        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto phi = hiprand_uniform(&d_state[i]) * M_PI;
    auto theta = hiprand_uniform(&d_state[i]) * 2 * M_PI;
    d_X[n].x = d_X[i].x + mean_distance / 4 * sinf(theta) * cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance / 4 * sinf(theta) * sinf(phi);
    d_X[n].u = d_X[i].u / 2;
    d_X[n].z = d_X[i].z + mean_distance / 4 * cosf(theta);
    d_X[i].u = d_X[i].u / 2;
    d_X[n].v = d_X[i].v / 2;
    d_X[i].v = d_X[i].v / 2;
    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = d_X[i].phi;
    d_type[n] = d_type[i];
}


int main(int argc, const char* argv[])
{
    // Initial state
    Solution<Cell, Grid_solver> cells(n_max);
    *cells.h_n = n_0;
    relaxed_sphere(0.75, cells);
    Property<n_max, Cell_types> type("type");
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    for (auto i = 0; i < n_0; i++) {
        type.h_prop[i] = mesenchyme;
    }
    cells.copy_to_device();
    type.copy_to_device();
    Property<n_max, int> n_mes_nbs("n_mes_nbs");
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    Property<n_max, int> n_epi_nbs("n_epi_nbs");
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));
    hiprandState* d_state;  // For proliferations
    hipMalloc(&d_state, n_max * sizeof(hiprandState));
    auto seed = time(NULL);
    setup_rand_states<<<(n_max + 128 - 1) / 128, 128>>>(n_max, seed, d_state);


    // Find epithelium
    thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_0, 0);
    cells.take_step<epi_turing_mes_noturing>(0);
    cells.copy_to_host();
    n_mes_nbs.copy_to_host();
    for (auto i = 0; i < n_0; i++) {
        if (n_mes_nbs.h_prop[i] < 20) {
            type.h_prop[i] = epithelium;
            auto dist = sqrtf(cells.h_X[i].x * cells.h_X[i].x +
                              cells.h_X[i].y * cells.h_X[i].y +
                              cells.h_X[i].z * cells.h_X[i].z);
            cells.h_X[i].theta = acosf(cells.h_X[i].z / dist);
            cells.h_X[i].phi = atan2(cells.h_X[i].y, cells.h_X[i].x);

            cells.h_X[i].u = rand() / (RAND_MAX + 1.) / 5 - 0.1;
            cells.h_X[i].v = rand() / (RAND_MAX + 1.) / 5 - 0.1;
        }
    }
    cells.copy_to_device();
    type.copy_to_device();

    // Integrate positions
    Vtk_output output("branching");
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        cells.copy_to_host();
        type.copy_to_host();

        std::thread calculation([&] {
            for (auto i = 0; i <= skip_steps; i++) {
                proliferate<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                    0.75, cells.d_X, cells.d_n, d_state);
                thrust::fill(thrust::device, n_mes_nbs.d_prop,
                    n_mes_nbs.d_prop + cells.get_d_n(), 0);
                thrust::fill(thrust::device, n_epi_nbs.d_prop,
                    n_epi_nbs.d_prop + cells.get_d_n(), 0);
                cells.take_step<epi_turing_mes_noturing>(dt);
            }
        });

        output.write_positions(cells);
        output.write_polarity(cells);
        output.write_field(cells, "u", &Cell::u);
        output.write_field(cells, "v", &Cell::v);
        output.write_property(type);

        calculation.join();
    }

    return 0;
}
