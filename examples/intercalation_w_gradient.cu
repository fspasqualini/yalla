#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/links.cuh"
#include "../include/meix.cuh"
#include "../include/polarity.cuh"
#include "../include/property.cuh"
#include "../include/solvers.cuh"
#include "../include/vtk.cuh"

const auto r_max = 1.0;
const auto r_min = 0.8;
const auto dt = 0.1f;
const auto n_max = 150000;
const auto prots_per_cell = 1;
const auto protrusion_strength = 0.2f;
const auto r_protrusion = 2.0f;
float mean_proliferation_rate = 0.015f;
int n_time_steps = 500;

enum Cell_types { mesenchyme, epithelium };

__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;
__device__ int* d_epi_nbs;

MAKE_PT(Cell, w, f, theta, phi);

__device__ Cell force(Cell Xi, Cell r, float dist, int i, int j)
{
    Cell dF{0};

    if (i == j) {
        dF.w = -0.01 * (d_type[i] == mesenchyme) * Xi.w;
        if (Xi.w < 0.f) Xi.w = 0.f;
        dF.f = -0.01 * (d_type[i] == mesenchyme) * Xi.f;
        if (Xi.f < 0.f) Xi.f = 0.f;

        return dF;
    }

    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        if (d_type[i] == mesenchyme)
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0);
        else
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0) * 2.f;
    } else {
        F = fmaxf(0.9 - dist, 0) * 2.f - fmaxf(dist - 0.9, 0) * 2.f;
    }
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    dF.w = -r.w * (d_type[i] == mesenchyme) * 0.1f;
    dF.f = -r.f * (d_type[i] == mesenchyme) * 0.1f;

    if (d_type[j] == epithelium)
        atomicAdd(&d_epi_nbs[i], 1);
    else
        atomicAdd(&d_mes_nbs[i], 1);

    if (Xi.w < 0.f) Xi.w = 0.f;
    if (Xi.f < 0.f) Xi.f = 0.f;
    if (d_type[i] == mesenchyme or d_type[j] == mesenchyme) return dF;

    dF += rigidity_force(Xi, r, dist) * 0.15;
    return dF;
}

__global__ void proliferate(float mean_rate, float mean_distance, Cell* d_X,
    int* d_n_cells, hiprandState* d_state)
{
    D_ASSERT(*d_n_cells * mean_rate <= n_max);
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *d_n_cells * (1 - mean_rate))
        return;  // Dividing new cells is problematic!

    switch (d_type[i]) {
        case mesenchyme: {
            return;
        }
        case epithelium: {
            if (d_epi_nbs[i] > 14) return;
            if (d_mes_nbs[i] < 1) return;
            auto rnd = hiprand_uniform(&d_state[i]);
            if (rnd > mean_rate) return;

        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto theta = hiprand_uniform(&d_state[i]) * 2 * M_PI;
    auto phi = hiprand_uniform(&d_state[i]) * M_PI;
    d_X[n].x = d_X[i].x + mean_distance / 4 * sinf(theta) * cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance / 4 * sinf(theta) * sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance / 4 * cosf(theta);
    if (d_type[i] == mesenchyme) {
        d_X[n].w = d_X[i].w / 2;
        d_X[i].w = d_X[i].w / 2;
    } else {
        d_X[n].w = d_X[i].w;
    }
    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = d_X[i].phi;
    d_type[n] = d_type[i];
}

__global__ void update_protrusions(const int n_cells,
    const Grid<n_max>* __restrict__ d_grid, const Cell* __restrict d_X,
    hiprandState* d_state, Link* d_link)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells * prots_per_cell) return;

    auto j = static_cast<int>((i + 0.5) / prots_per_cell);
    auto rand_nb_cube =
        d_grid->d_cube_id[j] +
        d_nhood[min(static_cast<int>(hiprand_uniform(&d_state[i]) * 27), 26)];
    auto cells_in_cube =
        d_grid->d_cube_end[rand_nb_cube] - d_grid->d_cube_start[rand_nb_cube];
    if (cells_in_cube < 1) return;

    auto a = d_grid->d_point_id[j];
    auto b =
        d_grid->d_point_id[d_grid->d_cube_start[rand_nb_cube] +
                           min(static_cast<int>(
                                   hiprand_uniform(&d_state[i]) * cells_in_cube),
                               cells_in_cube - 1)];
    D_ASSERT(a >= 0);
    D_ASSERT(a < n_cells);
    D_ASSERT(b >= 0);
    D_ASSERT(b < n_cells);
    if (a == b) return;

    if ((d_type[a] != mesenchyme) or (d_type[b] != mesenchyme)) return;

    auto link = &d_link[a * prots_per_cell + i % prots_per_cell];

    auto old_r = d_X[link->a] - d_X[link->b];
    auto old_dist = norm3df(old_r.x, old_r.y, old_r.z);
    auto new_r = d_X[a] - d_X[b];
    auto new_dist = norm3df(new_r.x, new_r.y, new_r.z);
    if (new_dist > r_protrusion) return;

    auto not_initialized = link->a == link->b;
    auto noise = hiprand_uniform(&d_state[i]);
    auto superficial = d_X[a].w + d_X[b].w > 0.3f;  //sort cells close to the w
    auto parallel_to_w_gradient = false;            //source
    auto normal_to_f_gradient = false;
    if (superficial) {  //cells close to the w source respond to the f gradient
        normal_to_f_gradient =
            fabs(new_r.f / new_dist) < fabs(old_r.f / old_dist) * (1.f - noise);
    } else {            //cells close to the f source respond to the w gradient
        parallel_to_w_gradient =
            fabs(new_r.w / new_dist) > fabs(old_r.w / old_dist) * (1.f - noise);
    }

    if (not_initialized or parallel_to_w_gradient or normal_to_f_gradient) {
        link->a = a;
        link->b = b;
    }
}


int main(int argc, char const* argv[])
{
    std::string ic_file_name = "examples/sphere_ic.vtk";

    // Load the initial conditions
    Vtk_input input(ic_file_name);
    int n0 = input.n_bolls;
    Solution<Cell, n_max, Grid_solver> sphere(n0);

    input.read_positions(sphere);
    input.read_polarity(sphere);

    Property<n_max, Cell_types> type;
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    Property<n_max, int> intype;

    input.read_property(intype, "cell_type");  // we read it as an int, then we
                                               //translate to enum "Cell_types"

    for (int i = 0 ; i < n0 ; i++) {
        sphere.h_X[i].w = 0.0f;
        if (intype.h_prop[i] == 0) {
            type.h_prop[i] = mesenchyme;
        } else if (intype.h_prop[i] == 1) {
            type.h_prop[i] = epithelium;
            if (sphere.h_X[i].z > 0.0f){
                sphere.h_X[i].w = 1.0f;
                if (sphere.h_X[i].x > 0.0f and abs(sphere.h_X[i].y) < 2.5f and
                    sphere.h_X[i].z < 3.0f)
                    sphere.h_X[i].f = 1.0f;
            }

        }
    }

    sphere.copy_to_device();
    type.copy_to_device();

    Property<n_max, int> n_mes_nbs("n_mes_nbs");
    Property<n_max, int> n_epi_nbs("n_epi_nbs");
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

    // Declaration of links
    Links<static_cast<int>(n_max * prots_per_cell)> protrusions(
        protrusion_strength, n0 * prots_per_cell);
    auto intercalation =
    std::bind(link_forces<static_cast<int>(n_max * prots_per_cell), Cell>,
        protrusions, std::placeholders::_1, std::placeholders::_2);

    Grid<n_max> grid;

    // State for links
    hiprandState* d_state;
    hipMalloc(&d_state, n_max * sizeof(hiprandState));
    auto seed = time(NULL);
    setup_rand_states<<<(n_max + 128 - 1) / 128, 128>>>(
        n_max, seed, d_state);

    Vtk_output output("intercalation_w_gradient", true);
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        sphere.copy_to_host();
        protrusions.copy_to_host();
        type.copy_to_host();

        thrust::fill(thrust::device, n_mes_nbs.d_prop,
            n_mes_nbs.d_prop + sphere.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + sphere.get_d_n(), 0);

        protrusions.set_d_n(sphere.get_d_n() * prots_per_cell);
        grid.build(sphere, r_protrusion);
        update_protrusions<<<(protrusions.get_d_n() + 32 - 1) / 32, 32>>>(
            sphere.get_d_n(), grid.d_grid, sphere.d_X,
            protrusions.d_state, protrusions.d_link);

        sphere.take_step<force>(dt, intercalation);

        proliferate<<<(sphere.get_d_n() + 128 - 1) / 128, 128>>>(
            mean_proliferation_rate, r_min, sphere.d_X,
            sphere.d_n, d_state);

        output.write_positions(sphere);
        output.write_links(protrusions);
        output.write_polarity(sphere);
        output.write_property(type);
        output.write_field(sphere);
        output.write_field(sphere, "f", &Cell::f);
    }

    return 0;
}
