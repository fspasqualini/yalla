#include "hip/hip_runtime.h"
// Simulate polarities aligining with gradient
#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/polarity.cuh"
#include "../include/solvers.cuh"
#include "../include/vtk.cuh"


const auto r_max = 1;
const auto D = 1;
const auto n_cells = 61;
const auto n_time_steps = 150;
const auto dt = 0.025;

MAKE_PT(Po_cell4, w, theta, phi);


__device__ Po_cell4 diffusion(Po_cell4 Xi, Po_cell4 r, float dist, int i, int j)
{
    Po_cell4 dF{0};
    if (i == j) return dF;

    if (dist > r_max) return dF;

    dF.w = i == 11 ? 0 : -r.w * D;
    if (r.w > 0) return dF;

    // U_WNT = - ΣXj.w*(n_i . r_ij/r)^2/2 to bias along w
    Polarity rhat{acosf(-r.z / dist), atan2(-r.y, -r.x)};
    dF += (Xi.w - r.w) * pcp_force(Xi, rhat);

    return dF;
}


int main(int argc, const char* argv[])
{
    // Prepare initial state
    Solution<Po_cell4, n_cells, Tile_solver> bolls;
    regular_hexagon(0.75, bolls);
    for (auto i = 0; i < n_cells; i++) {
        if (i == 11) {
            bolls.h_X[i].w = 50;
        } else {
            auto r = bolls.h_X[i] - bolls.h_X[11];  // Tilt polarities towards
            bolls.h_X[i].theta = 0.01;              // source to end w/ all
            bolls.h_X[i].phi = atan2(-r.y, -r.x);   // pointing the same way.
        }
    }
    bolls.copy_to_device();

    // Integrate cell positions
    Vtk_output output("wnt");
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        bolls.copy_to_host();
        bolls.take_step<diffusion>(dt);
        output.write_positions(bolls);
        output.write_polarity(bolls);
        output.write_field(bolls);
    }

    return 0;
}
